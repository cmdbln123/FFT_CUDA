#include "hip/hip_runtime.h"
#include "FFT_cooleytukey.cuh"
#define S_NUM 2

void ExeFft(int N1, int N2,float2* h_dataI, float2* h_dataO, int k)
{
	hipEvent_t start_t, stop_t;
	float costtime=0;
	float2 *d_dataI;
	float2 *d_dataO;
	float2 *round_one;
	float2 *round_two;
	float2 *h_dataO_temp;
	round_one = (float2 *)malloc(N1*N2*sizeof(float2));
	round_two = (float2 *)malloc(N2*N1*sizeof(float2));
	float2 *d_round_one;
	float2 *d_round_two;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_round_one, N1*S_NUM*sizeof(float2)));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dataO, N1*S_NUM*sizeof(float2)));
	unsigned int timer = 0;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	hipStream_t* cudastream;

	Transform(h_dataI, round_one, N2, N1);

	cudastream = new hipStream_t[S_NUM];
	for (int i=0;i<N2;i++)
	{
		CUDA_SAFE_CALL(hipStreamCreate(cudastream+i%S_NUM));
		CUDA_SAFE_CALL(hipMemcpyAsync(d_round_one + (i%S_NUM)*N1, round_one + i*N1, N1*sizeof(float2), hipMemcpyHostToDevice, cudastream[i%S_NUM]));
		DoFft(N1, 1, d_round_one + (i%S_NUM)*N1, d_dataO + (i%S_NUM)*N1 , k, cudastream[i%S_NUM]);
		CUDA_SAFE_CALL(hipMemcpyAsync(round_one + i*N1, d_dataO + (i%S_NUM)*N1, sizeof(float2)*N1, hipMemcpyDeviceToHost, cudastream[i%S_NUM]));
	}
	hipDeviceSynchronize();

	delete[] cudastream;
	CUDA_SAFE_CALL(hipFree(d_round_one));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_round_two, N2*S_NUM*sizeof(float2)));
	CUDA_SAFE_CALL(hipFree(d_dataO));
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dataO, N2*S_NUM*sizeof(float2)));

	Transform(round_one, round_two, N1, N2);
	whirl_factor(round_two, round_two, N1, N2);
	h_dataO_temp = round_one;
	cudastream = new hipStream_t[S_NUM];
	for (int i=0;i<N1;i++)
	{
		CUDA_SAFE_CALL(hipStreamCreate(cudastream+i%S_NUM));
		CUDA_SAFE_CALL(hipMemcpyAsync(d_round_two+ (i%S_NUM)*N2, round_two + i*N2, N2*sizeof(float2), hipMemcpyHostToDevice, cudastream[i%S_NUM]));
		DoFft(N2, 1, d_round_two + (i%S_NUM)*N2, d_dataO + (i%S_NUM)*N2, k, cudastream[i%S_NUM]);	
		CUDA_SAFE_CALL(hipMemcpyAsync(h_dataO_temp + i*N2, d_dataO + (i%S_NUM)*N2, N2*sizeof(float2), hipMemcpyDeviceToHost, cudastream[i%S_NUM]));
	}
	hipDeviceSynchronize();//ͬ����������֤������һ����
	cutStopTimer(timer);
	float t = cutGetTimerValue(timer);
	cutDeleteTimer(timer);
	cout << "ct time:\t" << t << "\t" << N1 << "\t" << N2<< endl;

	for (int i=0;i<N1;i++)
	{
		for (int j=0;j<N2;j++)
		{
			h_dataO[j*N1+i] = h_dataO_temp[i*N2+j];
		}
	}
	delete[] cudastream;
	delete[] round_one;
	delete[] round_two;
	CUDA_SAFE_CALL(hipFree(d_round_two));
}

void DoFft(int N, int cN,float2* dataI, float2* dataO, int k, hipStream_t cudastream)
{
	//assert(pow(2,log(1.0*N)/log(2.0))!=N);
	int R = 2;
	int T = (N/R<THREAD_X)? N/R:THREAD_X;
	int BX = ((N-1)/(R*T) + 1<65536)? ((N-1)/(R*T) + 1):65536;
	int BY =  cN*(-1)*(k - 1)*M_FACTOR/2;//Ϊ��С���任�ı�ͨ���� ���ʽ
	if (BY==0){
		BY = cN;
	}
	dim3 dimgrid(BX,BY,1);
	dim3 dimblock(T,1,1);
	float2* temp = dataO;
	float2* dataIn = dataI;

	unsigned int timer = 0;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	int nb,nt;
	nt = 512;
	nb = N/1024;
	sorting<<<nb,nt, 0,cudastream>>>(dataIn, dataIn, N);
	hipDeviceSynchronize();//ͬ����������֤������һ����
	cutStopTimer(timer);
	float t = cutGetTimerValue(timer);
	cutDeleteTimer(timer);
	cout << "\tct part1\t" << t << "\t";

	cutCreateTimer(&timer);
	cutStartTimer(timer);
	for (int Ns=1; Ns<N; Ns*=R)
	{
		GPU_FFT_cooleytukey<<<dimgrid, dimblock, 0, cudastream>>>(N, R, Ns, dataIn, temp, k);
		hipDeviceSynchronize();

		float2 *change;
		change = temp;
		temp = dataIn;
		dataIn = change;		
	}

	if (dataIn != dataO)
	{
		hipMemcpyAsync(dataO,dataIn,sizeof(float2)*N*BY, hipMemcpyDeviceToDevice, cudastream);
	}
	hipDeviceSynchronize();//ͬ����������֤������һ����
	cutStopTimer(timer);
	t = cutGetTimerValue(timer);
	cutDeleteTimer(timer);
	cout << "\tct part2\t" << t << "\n";
}


__global__ void GPU_FFT_cooleytukey(int N, int R, int Ns,float2* dataI, float2* dataO ,int k)
{
	int b, T, t;
	b = blockIdx.x;
	T =  blockDim.x;
	t = threadIdx.x;
	int j  = (blockIdx.x)*T + t; 
	if (j< N/R)
	{
		FftIteration(j, N, R, Ns, dataI+blockIdx.y*N, dataO+blockIdx.y*N, k);		
	}

}



void Transform(float2* dataIn, float2* dataOut, int N1, int N2)
	//float2* d_dataIn, float2* d_dataOut, int N1, int N2
{
	//Fermi�ܹ�CUDA������Ż�
	//Whitepaper
	//NVIDIA��s Next Generation
	//	CUDATM Compute Architecture:
	//	FermiTM

	//dim3 threads(16,16,1);
	//dim3 blocks(1,1,1);
	//blocks.x = (N1+threads.x-1)/threads.x;
	//blocks.y = (N2+threads.y-1)/threads.y;
	//Trans<<<blocks, threads>>>(d_dataIn, d_dataOut, N1, N2);
	int idin;
	int idout;
	for (int i=0;i<N1;i++)
	{
		for (int j=0;j<N2;j++)
		{
			idin = j*N1+i;
			idout = i*N2+j;
			dataOut[idout].x = dataIn[idin].x;
			dataOut[idout].y = dataIn[idin].y;
		}
	}
	return ;
}

void whirl_factor(float2* dataI, float2 *dataO, int N1, int N2)
{
	//A4(k,j) = cos(2*pi*(k-1)*(j-1)/(n1*n2))-(-1)^0.5*sin(2*pi*(k-1)*(j-1)/(n1*n2));
	//dim3 blocks;
	//dim3 threads;
	int index;
	float id_x;
	float id_y;
	for (int i=0;i<N1;i++)
	{
		for (int j=0;j<N2;j++)
		{
			id_x = i;
			id_y = j;
			index = id_x * N2 + id_y;
			float2 w;
			float an = -2.0*M_PI*(id_x)*(id_y)/(N1*N2);
			w.x = cos(an);
			w.y = sin(an);
			dataO[index] = h_multi(w, dataI[index]);
		}
	}

}

float2 h_multi(float2 a, float2 b)
{
	float2 c;

	c.x = a.x*b.x - a.y*b.y;
	c.y = a.x*b.y + a.y*b.x;
	return c;
}

__device__ int turn_round(int a, int len)
{
	int x,y;
	x = 1;
	y = 0;
	for (int i=0;i<len;i++)
	{
		int temp;
		temp = a&x;
		temp = temp >> i;
		temp = temp << (len-1-i);
		y |= temp;
		x = x << 1;
	}
	return y;
}

__global__ void sorting(float2 *dataI, float2 *dataO, int len)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int index = tid + bid* blockDim.x;
	int totalthread  = blockDim.x * gridDim.x;
	int a, b, n;
	n = log(1.0*len)/log(2.0);
	while (index<len)
	{
		a = index;
		b = turn_round(index, n);
		if (a<=b)
		{
			float2 tempa,tempb;
			tempa = dataI[a];
			tempb = dataI[b];
			dataO[a] = tempb;
			dataO[b] = tempa;
		}

		index += totalthread;
	}
	return ;
}

__device__ 
	void FFT_2(float2* v)
{
	float2 v0;
	v0.x = v[0].x;
	v0.y = v[0].y;

	v[0].x = v0.x + v[1].x;
	v[0].y = v0.y + v[1].y;

	v[1].x = v0.x - v[1].x;
	v[1].y = v0.y - v[1].y;
}

__device__ 
void FftIteration(int j, int N, int R, int Ns, float2* data0, float2* data1, int k)
{
	float2 v[2];
	int b = blockIdx.x;
	int t = threadIdx.x;
	int T = blockDim.x;
	int ns;
	int idxD;
	int idxS;
	ns = Ns;
	idxS = (j/ns)*R*ns + j%ns;
	float angle = -2*M_PI*(j%ns)/(ns*R);
	for (int r= 0;r<R;r++)
	{
		v[r].x = data0[idxS + r*ns].x;
		v[r].y = k*data0[idxS + r*ns].y;

		float2 temp;
		temp = v[r];

		v[r].x = temp.x*__cosf(angle*r) - temp.y*__sinf(angle*r);
		v[r].y = temp.y*__cosf(angle*r) + temp.x*__sinf(angle*r);
	}
	FFT_2(v);

	idxD = idxS;
	for (int r=0;r<R;r++)
	{
		data1[idxD+r*ns].x = v[r].x;
		data1[idxD+r*ns].y = k*v[r].y;
	}
	return ;
}
